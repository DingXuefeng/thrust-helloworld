#include "hip/hip_runtime.h"
/*****************************************************************************/
// Author: Xuefeng Ding <dingxf@ihep.ac.cn> @ IHEP-CAS
//
// Date: 2023 April 30
// Version: v1.0
// Description: Thrust hello world
//
// All rights reserved. 2023 copyrighted.
/*****************************************************************************/
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <chrono>
#ifdef USE_FLOAT
using real_t = float;
#else
using real_t = double;
#endif
struct conv : public thrust::unary_function<int, real_t>
{
    conv(real_t *x, real_t *fx,
         int task_a, int task_b, int task_c,
         int output) : m_x(x), m_fx(fx),
                       m_tasksize{task_a, task_b, task_c},
                       m_taskshift{0, task_a, task_a + task_b},
                       m_output(output) {}
    void update(
        real_t lightYield, real_t sigma)
    {
        m_lightYield = lightYield;
        m_sigmaInv = 1. / sigma;
    }
    __device__
        real_t
        operator()(const int id)
    {
        const int task_id = id / m_output;
        const int output_id = id % m_output;
        const real_t *const x = m_x + m_taskshift[task_id];
        const real_t *const fx = m_fx + m_taskshift[task_id];

        const real_t y = output_id;

        real_t sum = 0;
        for (int i = 0; i < m_tasksize[task_id]; ++i)
        {
            constexpr real_t inv_sqrt_2pi = 0.3989422804014327;
            const real_t tmp = (y - x[i] * m_lightYield) * m_sigmaInv;
            constexpr real_t half = 0.5;
            const real_t detector = inv_sqrt_2pi * exp(-tmp * tmp * half) * m_sigmaInv;
            sum += fx[i] * detector;
        }
        return sum;
    }

    const real_t *const m_x;
    const real_t *const m_fx;
    const int m_tasksize[3];
    const int m_taskshift[3];
    const int m_output;

    real_t m_lightYield;
    real_t m_sigmaInv;
};

class Task
{
public:
    Task(int task_a, int task_b, int task_c,
         int output,
         real_t *x, real_t *fx) : m_task_a(task_a), m_task_b(task_b), m_task_c(task_c),
                                  m_x(task_a + task_b + task_c),
                                  m_fx(task_a + task_b + task_c),
                                  m_output(output),
                                  m_y(output * 3),
                                  op(thrust::raw_pointer_cast(m_x.data()),
                                     thrust::raw_pointer_cast(m_fx.data()),
                                     task_a, task_b, task_c,
                                     output)
    {
        thrust::copy(x, x + task_a + task_b + task_c, m_x.begin());
        thrust::copy(fx, fx + task_a + task_b + task_c, m_fx.begin());
    }
    void run(real_t lightYield, real_t sigma)
    {
        op.update(lightYield, sigma);
        thrust::transform(thrust::counting_iterator<int>(0),
                          thrust::counting_iterator<int>(m_y.size()),
                          m_y.begin(),
                          op);
    }
    void output()
    {
        thrust::host_vector<real_t> y = m_y;
        std::vector<real_t> y_sum(m_output);
        for (int i = 0; i < m_output; ++i)
            for (int j = 0; j < 5; ++j)
                y_sum[i] += y[i + j * m_output];
        for (int i = 0; i < 10; ++i)
            std::cout << i << " " << y_sum[i] << std::endl;
    }

private:
    const int m_task_a, m_task_b, m_task_c;
    const int m_output;
    thrust::device_vector<real_t> m_x;
    thrust::device_vector<real_t> m_y;
    thrust::device_vector<real_t> m_fx;
    conv op;
};

int main()
{
    // prepare input
    constexpr int task_a = 1000, task_b = 200, task_c = 4500;
    constexpr int total = task_a + task_b + task_c;
    constexpr int output = 1000;
    real_t x[total], fx[total];
    for (int i = 0; i < task_a; ++i)
    {
        x[i] = 1.0 * i / task_a;
        fx[i] = 5.3 * exp(-x[i] / 0.3);
    }
    for (int i = 0; i < task_b; ++i)
    {
        x[task_a + i] = 0.2 * i / task_b;
        fx[task_a + i] = 5.3 * exp(-x[task_a + i] / 0.03);
    }
    for (int i = 0; i < task_c; ++i)
    {
        x[task_a + task_b + i] = 1.5 * i / task_c;
        fx[task_a + task_b + i] = 5.3 * x[task_a + task_b + i] * (1.5 - x[task_a + task_b + i]);
    }
    // pass to device
    Task hi(task_a, task_b, task_c, output, x, fx);

    auto wall_t0 = std::chrono::high_resolution_clock::now();
    hi.run(500, 20);
    auto wall_t1 = std::chrono::high_resolution_clock::now();
    auto wall_t = std::chrono::duration<real_t, std::milli>(wall_t1 - wall_t0).count();
    std::cout << __LINE__ << "  " << wall_t << " ms" <<std::endl;

    hi.output();
    return 0;
}