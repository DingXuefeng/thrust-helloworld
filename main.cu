#include "hip/hip_runtime.h"
/*****************************************************************************/
// Author: Xuefeng Ding <dingxf@ihep.ac.cn> @ IHEP-CAS
//
// Date: 2023 April 30
// Version: v1.0
// Description: Thrust hello world
//
// All rights reserved. 2023 copyrighted.
/*****************************************************************************/
#include <thrust/functional.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/transform.h>
#include <nvToolsExt.h>
#include <chrono>
#include <vector>
#ifdef USE_FLOAT
using real_t = float;
#else
using real_t = double;
#endif

struct conv : public thrust::unary_function<int, real_t>
{
    conv(int *nInput,
         real_t **x,
         real_t **fx,
         int output)
        : m_N(nInput),
          m_x(x),
          m_fx(fx),
          m_output(output) {}
    void update(
        real_t lightYield, real_t sigma)
    {
        m_lightYield = lightYield;
        m_sigmaInv = 1. / sigma;
    }
    __device__
        real_t
        operator()(const int id)
    {
        const int task_id = id / m_output;
        const int output_id = id % m_output;

        const real_t my = -1.0 * output_id;

        real_t sum = 0;
        for (int i = 0; i < m_N[task_id]; ++i)
        {
            constexpr real_t inv_sqrt_2pi = 0.3989422804014327;
            const real_t tmp = (m_x[task_id][i] * m_lightYield + my) * m_sigmaInv;
            constexpr real_t mhalf = -0.5;
            const real_t detector = inv_sqrt_2pi * exp(mhalf * tmp * tmp) * m_sigmaInv;
            sum += m_fx[task_id][i] * detector;
        }
        return sum;
    }

    int *const m_N;
    real_t **const m_x;
    real_t **const m_fx;
    const int m_output;

    real_t m_lightYield;
    real_t m_sigmaInv;
};

class Task
{
public:
    Task(const std::vector<std::vector<real_t>> &x,
         const std::vector<std::vector<real_t>> &fx,
         int output) : m_N(x.size()),
                       m_x(x.size()),
                       m_fx(x.size()),
                       m_output(output),
                       m_y(output * x.size()),
                       op(thrust::raw_pointer_cast(m_N.data()),
                          thrust::raw_pointer_cast(m_x.data()),
                          thrust::raw_pointer_cast(m_fx.data()),
                          output)
    {
        for (int i = 0; i < x.size(); ++i)
        {
            m_N[i] = x[i].size();
            m_x_store.emplace_back(x[i]);
            m_x[i] = thrust::raw_pointer_cast(m_x_store[i].data());
            m_fx_store.emplace_back(fx[i]);
            m_fx[i] = thrust::raw_pointer_cast(m_fx_store[i].data());
        }
    }
    void run(real_t lightYield, real_t sigma)
    {
        op.update(lightYield, sigma);
        nvtxRangePushA("transform");
        thrust::transform(thrust::counting_iterator<int>(0),
                          thrust::counting_iterator<int>(m_y.size()),
                          m_y.begin(),
                          op);
        nvtxRangePop();
    }
    void output()
    {
        thrust::host_vector<real_t> y = m_y;
        std::vector<real_t> y_sum(m_output);
        for (int i = 0; i < m_output; ++i)
            for (int j = 0; j < 5; ++j)
                y_sum[i] += y[i + j * m_output];
        for (int i = 0; i < 10; ++i)
            std::cout << i << " " << y_sum[i] << std::endl;
    }

private:
    const int m_output;
    thrust::device_vector<int> m_N;
    std::vector<thrust::device_vector<real_t>> m_x_store;
    std::vector<thrust::device_vector<real_t>> m_fx_store;
    thrust::device_vector<real_t *> m_x;
    thrust::device_vector<real_t *> m_fx;
    thrust::device_vector<real_t> m_y;
    conv op;
};

int main()
{
    // prepare input
    constexpr int task_a = 1000, task_b = 200, task_c = 4500;
    std::vector<std::vector<real_t>> x(3), fx(3);
    constexpr int output = 19315;
    for (int i = 0; i < task_a; ++i)
    {
        x[0].push_back(1.0 * i / task_a);
        fx[0].push_back(5.3 * exp(-x[0][i] / 0.3));
    }
    for (int i = 0; i < task_b; ++i)
    {
        x[1].push_back(0.2 * i / task_b);
        fx[1].push_back(5.3 * exp(-x[1][i] / 0.03));
    }
    for (int i = 0; i < task_c; ++i)
    {
        x[2].push_back(1.5 * i / task_c);
        fx[2].push_back(5.3 * x[2][i] * (1.5 - x[2][i]));
    }
    // pass to device
    Task hi(x, fx, output);

    auto wall_t0 = std::chrono::high_resolution_clock::now();
    hi.run(500 * 19315 / 1000, 20);
    auto wall_t1 = std::chrono::high_resolution_clock::now();
    auto wall_t = std::chrono::duration<real_t, std::milli>(wall_t1 - wall_t0).count();
    std::cout << __LINE__ << "  " << wall_t << " ms" << std::endl;

    hi.output();
    return 0;
}
