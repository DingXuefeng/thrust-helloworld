#include <iostream>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);

    if (deviceCount == 0) {
        std::cerr << "No CUDA devices found." << std::endl;
        return 1;
    }

    for (int device = 0; device < deviceCount; ++device) {
        hipDeviceProp_t deviceProp;
        hipGetDeviceProperties(&deviceProp, device);

        std::cout << "Device " << device << ": "
                  << deviceProp.name << std::endl
                  << "  Compute Capability: " << deviceProp.major << "." << deviceProp.minor
                  << std::endl;
    }

    return 0;
}

